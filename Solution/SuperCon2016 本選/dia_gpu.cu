#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/time.h>
#include <stdlib.h>
#include <assert.h>

#include "input.h"

unsigned long seed[4];

int G[MAX_N][MAX_N]; //隣接グラフ

void rnd_init(unsigned int s){
	for(int i = 1; i <= 4; ++i)seed[i - 1] = s = 1812433253U * (s ^ (s >> 30)) + i;
}

unsigned long xor128(void) {
    unsigned long t;
	t=(seed[0]^(seed[0]<<11));seed[0]=seed[1];seed[1]=seed[2];seed[2]=seed[3]; return( seed[3]=(seed[3]^(seed[3]>>19))^(t^(t>>8)) );
}
int rnd(int k) {
    return xor128() % k;
}

void swap(int *a, int *b) {
    int t = *a;
    *a = *b;
    *b = t;
}

void random_shuffle(int a[], int size) {
    int i, r;
    for (i = size; i > 0; i--) {
        r = rnd(i);
        swap(a+r, a+i-1);
    }
}

int suitable(int potential_edges[], int n) {
    int i, j, flag = 0;
    for (i = 0; i < n; i++) {
        if (potential_edges[i] > 0) {
            flag = 1;
            break;
        }
    }
    if (!flag) return 1;

    for (i = 0; i < n; i++) {
        for (j = i+1; j < n; j++) {
            if (potential_edges[i] > 0 && potential_edges[j] > 0 && G[i][j] == 0)
                return 1;
        }
    }
    return 0;
}

int try_creation(int n, int d) {
    int stubs[n*d];
    int potential_edges[n];
    int size = n * d;

    memset(G, 0, sizeof(G));
    int i, j;
    for (i = 0; i < d; i++)
        for (j = 0; j < n; j++)
            stubs[i * n + j] = j;

    int s1, s2;
    while (size > 0) {
        memset(potential_edges, 0, sizeof(potential_edges));
        random_shuffle(stubs, size);
        for (i = 0; i < size; i+=2) {
            s1 = stubs[i]; s2 = stubs[i+1];
            if (s1 != s2 && G[s1][s2] == 0) {
                G[s1][s2] = G[s2][s1] = 1;
            } else {
                potential_edges[s1]++;
                potential_edges[s2]++;
            }
        }

        if (!suitable(potential_edges, n)) return 0;

        size = 0;
        for (i = 0; i < n; i++) {
            for (j = 0; j < potential_edges[i]; j++)
                stubs[size++] = i;
        }
    }
    return 1;
}

int generate_random_graph(int n, int d) {
    if ((n * d) % 2 != 0 || !(0 < d && d < n)) return 0;
    while (!try_creation(n, d));
    return 1;
}

typedef struct{
	int to, dist;
}P;

__device__ P maxP(P l, P r){
	if(l.dist < r.dist)return r;
	else return l;
}

P maxP_C(P l, P r){
	if(l.dist < r.dist)return r;
	else return l;
}

P graph[MAX_N][16]; //隣接リスト
int graphptr[MAX_N];

#define CUDA_SAFE_CALL(func) \
do { \
    hipError_t err = (func); \
    if (err != hipSuccess) { \
        fprintf(stderr, "[Error] %s (error code: %d) at %s line %d\n", hipGetErrorString(err), err, __FILE__, __LINE__); \
        exit(err); \
    } \
} while(0)

typedef struct {
	int prior;
	P data;
} node_t;

typedef struct {
	node_t* nodes;
	int len;
	int size;
} heap_t;

__device__ node_t* realloc_g(node_t* ptr, int old_size, int new_size) {
	node_t* old = ptr;
	
	ptr = (node_t*)malloc(sizeof(node_t) * new_size);
	
	assert(ptr != NULL);
	if(old_size)
		memcpy(ptr, old, sizeof(node_t) * old_size);
	
	free(old);
	
	return ptr;
}

__device__ heap_t* calloc_g(int size) {
	heap_t* ptr = (heap_t*)malloc(size * sizeof(heap_t));
	
	assert(ptr != NULL);
	memset(ptr, 0, sizeof(heap_t) * size);
	
	return ptr;
}

__device__ void push(heap_t* h, int prior, P data) {
	if (h->len + 1 >= h->size) {
		int old = h->size;
		h->size = h->size ? h->size * 2 : 4;
		h->nodes = realloc_g(h->nodes, old, h->size);
	}
	int i = h->len + 1;
	int j = i / 2;
	
	while(i > 1 && h->nodes[j].prior > prior) {
		h->nodes[i] = h->nodes[j];
		i = j;
		j /= 2;
	}
	
	h->nodes[i].prior = prior;
	h->nodes[i].data = data;
	++h->len;
}

__device__ P pop(heap_t* h) {
	int i, j, k;
	if(!h->len) {
		P p;
		p.to = -1, p.dist = -1;
		return p;
	}
	
	P data = h->nodes[1].data;
	h->nodes[1] = h->nodes[h->len];
	--h->len;
	i = 1;
	
	while(true) {
		k = i;
		j = 2 * i;
		
		if(j <= h->len && h->nodes[j].prior < h->nodes[k].prior)
			k = j;
		
		if (j + 1 <= h->len && h->nodes[j + 1].prior < h->nodes[k].prior)
			k = j + 1;
		
		if(k == i)
			break;
		
		h->nodes[i] = h->nodes[k];
		i = k;
	}
	
	h->nodes[i] = h->nodes[h->len + 1];
	
	return data;
}

__device__ void dijkstra(int k, int *cost, int n, int d, P* graph) {
	heap_t* h = calloc_g(1);
	
	for(int i = 0; i < n; ++i) cost[i] = INF;
	
	cost[k] = 0;
	
	P t;
	
	t.to = k, t.dist = 0;
	push(h, t.dist, t);
	
	while(h->len) {
		P p = pop(h);
		
		int v = p.to;
		if(cost[v] < p.dist)
			continue;
		
		for(int i = 0; i < d; ++i) {
			P e = graph[v * MAX_D + i];
			
			if(cost[e.to] > cost[v] + e.dist) {
				cost[e.to] = cost[v] + e.dist;
				P m;
				m.to = e.to, m.dist = cost[e.to];
				
				push(h, m.dist, m);
			}
		}
	}
	
	free(h);
}

__device__ int AtreeDim(P* ret, P* graph, int n, int d) {
	int* cost = (int*)malloc(sizeof(int) * MAX_N);
	
	assert(cost != NULL);
	
	dijkstra(0, cost, n, d, graph);
	
	int maxV = -INF, maxI;
	for(int i = 0; i < n; ++i) {
		if(cost[i] > maxV) maxV = cost[i], maxI = i;
	}
	
	dijkstra(maxI, cost, n, d, graph);
	
	ret->to = maxI;
	maxV = -INF;
	
	for(int i = 0; i < n; ++i) {
		if(cost[i] > maxV) maxV = cost[i], maxI = i;
	}
	
	ret->dist = maxI;
	
	free(cost);
	
	return maxV;
}

__device__ P dfs(int v, int d, bool* used, P* graph){
	used[v] = true;
	
	P ret;
	ret.to = v; ret.dist = 0;
	for(int i = 0; i < d; ++i){
		int u = graph[v * MAX_D + i].to;
		int w = graph[v * MAX_D + i].dist;
		if(used[u])continue;
		P tmp = dfs(u, d, used, graph);
		P tmp2;
		tmp2.to = tmp.to; tmp2.dist = w + tmp.dist;
		ret = maxP(ret, tmp2);
	}
	return ret;
}

__device__ int treeDim(int n, int d, P* graph, P* ret){
	bool* used = (bool*)malloc(sizeof(bool) * n);
	
	for(int i = 0; i < n; ++i)used[i] = false;
	int v = dfs(0, d, used, graph).to;
	for(int i = 0; i < n; ++i)used[i] = false;
	P p = dfs(v, d, used, graph);
	ret->to = v, ret->dist = p.to;
	
	free(used);
	return p.dist;
}

__device__ int getI(int p, int c, int *N){
	int sum = 0;
	for(int i = 0; i < c; ++i){
		sum += N[i];
		if(p < sum)return i;
	}
	return -1;
}

int getI_C(int p){
	int sum = 0;
	for(int i = 0; i < c; ++i){
		sum += N[i];
		if(p < sum)return i;
	}
	return -1;
}

bool used_C[MAX_N];
P dfs_C(int v){
	used_C[v] = true;
	
	P ret;
	ret.to = v; ret.dist = 0;
	for(int i = 0; i < d; ++i){
		int u = graph[v][i].to;
		int w = graph[u][i].dist;
		if(used_C[u])continue;
		P tmp = dfs_C(u);
		P tmp2;
		tmp2.to = tmp.to; tmp2.dist = w + tmp.dist;
		ret = maxP_C(ret, tmp2);
	}
	return ret;
}

int treeDim_C(P* ret){
	for(int i = 0; i < n; ++i)used_C[i] = false;
	int v = dfs_C(0).to;
	for(int i = 0; i < n; ++i)used_C[i] = false;
	P p = dfs_C(v);
	ret->to = v, ret->dist = p.to;
	return p.dist;
}

__device__ void Gswap(int p, int q, P *graph, int n, int d, int c, int *W, int *N){
	bool* use = (bool*)malloc(sizeof(bool) * n);
	assert(use != NULL);
	memset(use, 0, sizeof(bool) * n);
	
	int* change = (int*)malloc(sizeof(int) * 250);
	assert(change != NULL);
	int chSize = 0;
	
	change[chSize++] = p;
	change[chSize++] = q;
	use[p] = true;
	use[q] = true;
	
	int tmp[MAX_D];
	for(int i = 0; i < d; ++i)tmp[i] = graph[p * MAX_D + i].to;
	
	for(int i = 0; i < d; ++i){
		graph[p * MAX_D + i].to = graph[q * MAX_D + i].to;
		
		for(int j = 0; j < d; ++j){
			int v = graph[p * MAX_D + j].to;
			if(!use[v])change[chSize++] = v, use[v] = true;
		}
	}
	
	for(int i = 0; i < d; ++i){
		graph[q * MAX_D + i].to = tmp[i];
		
		for(int j = 0; j < d; ++j){
			int v = graph[q * MAX_D + j].to;
			if(!use[v])change[chSize++] = v, use[v] = true;
		}	
	}
	
	for(int i = 0; i < d; ++i)graph[p * MAX_D + i].dist = W[getI(p, c, N) * MAX_C + getI(graph[p * d + i].to, c, N)];
	for(int i = 0; i < d; ++i)graph[q * MAX_D + i].dist = W[getI(q, c, N) * MAX_C + getI(graph[q * d + i].to, c, N)];
	
	for(int i = 0; i < chSize; ++i){
		int v = change[i];
		use[v] = false;
		for(int j = 0; j < d; ++j){
			if(graph[v * MAX_D + j].to == p)graph[v * MAX_D + j].to = q, graph[v * MAX_D + j].dist = W[getI(v, c, N) * MAX_C + getI(q, c, N)];
			else if(graph[v * MAX_D + j].to == q)graph[v * MAX_D + j].to = p, graph[v * MAX_D + j].dist = W[getI(v, c, N) * MAX_C + getI(p, c, N)];
		}
	}
	
	free(use);
	free(change);
}

bool use[MAX_N];
int change[100];
void Gswap_C(int p, int q){
	int chSize = 0;
	
	change[chSize++] = p;
	change[chSize++] = q;
	use[p] = true;
	use[q] = true;
	
	int tmp[MAX_D];
	for(int i = 0; i < d; ++i)tmp[i] = graph[p][i].to;
	
	for(int i = 0; i < d; ++i){
		graph[p][i].to = graph[q][i].to;
		
		for(int j = 0; j < d; ++j){
			int v = graph[p][i].to;
			if(!use[v])change[chSize++] = v, use[v] = true;
		}
	}
	for(int i = 0; i < d; ++i){
		graph[q][i].to = tmp[i];
		
		for(int j = 0; j < d; ++j){
			int v = graph[q][i].to;
			if(!use[v])change[chSize++] = v, use[v] = true;
		}
	}
	
	for(int i = 0; i < d; ++i)graph[p][i].dist = W[getI_C(p)][getI_C(graph[p][i].to)];
	for(int i = 0; i < d; ++i)graph[q][i].dist = W[getI_C(q)][getI_C(graph[q][i].to)];
	
	for(int i = 0; i < chSize; ++i){
		int v = change[i];
		use[v] = false;
		for(int j = 0; j < d; ++j){
			if(graph[v][j].to == p)graph[v][j].to = q, graph[v][j].dist = W[getI_C(v)][getI_C(q)];
			else if(graph[v][j].to == q)graph[v][j].to = p, graph[v][j].dist = W[getI_C(v)][getI_C(p)];
		}
	}
}

int ans[MAX_N * 16];

void debug(void){
	printf("\n");
	for(int i = 0; i < n; ++i){
		for(int j = 0; j < graphptr[i]; ++j){
			printf("CPU:i: %d to: %d dist: %d\n", i, graph[i][j].to, graph[i][j].dist);
		}
	}
	printf("\n");
}

void move(void){
	int ansSize = 0;
	for(int i = 0; i < n; ++i){
		for(int j = 0; j < d; ++j){
				if(graph[i][j].to > i){
				ans[ansSize++] = i;
				ans[ansSize++] = graph[i][j].to;
			}
		}
	}
}

int get_elapsed_time(struct timeval *begin, struct timeval *end)
{
	return (end->tv_sec - begin->tv_sec) * 1000
	+ (end->tv_usec - begin->tv_usec) / 1000;
}

__global__ void mm_main(int n, int d, int c, int *W, int* N, P v, P* graph, int* minVarr, P* stub){
	int i;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(i == v.to || i >= n)return;
	
	stub += i * MAX_N * MAX_D;
	memcpy(stub, graph, sizeof(P) * MAX_N * MAX_D);
	
	for(int i = 0; i < n; ++i){
		for(int j = 0; j < d; ++j){
			//printf("i: %d to: %d c:%d\n", i, graph[i * MAX_D + j].to, graph[i * MAX_D + j].dist);
		}
	}
	
	Gswap(v.to, i, stub, n, d, c, W, N);
	
	P m;
	int ret = AtreeDim(&m, stub, n, d);
	//printf("ret %d\n", ret);
	
	minVarr += i;
	*minVarr = ret;
	for(int i = 0; i < n; ++i){
		for(int j = 0; j < d; ++j){
		//	printf("SWAP: i: %d to: %d c:%d\n", i, stub[i * MAX_D + j].to, stub[i * MAX_D + j].dist);
		}
	}
	
	//printf("ret: %d\n", ret);
	
	Gswap(v.to, i, stub, n, d, c, W, N);
}

__global__ void initMinVArr(unsigned int n, int* minVarr) {
	unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
	
	if(x >= n) {
		return;
	}
	
	minVarr[x] = INF;
}

__global__ void getV(int n, int d, int* minVarr, P* stub) {
	int minV = INF;
	int minVidx = -1;
	
	for(int i = 0; i < n; ++i) {
		if(minV > minVarr[i]) {
			minVidx = i;
			minV = minVarr[i];
		}
	}
	
	P* rstub = stub;
	stub += minVidx * sizeof(P) * MAX_N * MAX_D;
	for(int i = 0; i < n * d; ++i) {
		rstub[i] = stub[i];
	}
	minVarr[0] = minVarr[minVidx];
}

int main(void) {
	input();
	
	int allMin = INF;
	
	const int BigCnt = 1, smallCnt = 1;
	
	//struct timeval t1, t2, t3, t4;
	//int ms = 0, ms2 = 0;
	
	int *WD, *ND;
	P* graphD;
		
	int *minVarr;
	P *stub;
		
	hipMalloc((void**)&WD, sizeof(int) * MAX_C * MAX_C);
	hipMalloc((void**)&ND, sizeof(int) * MAX_C);
	hipMalloc((void**)&graphD, sizeof(P) * MAX_N * MAX_D);
	hipMalloc((void**)&minVarr, sizeof(int) * BS);
	hipMalloc((void**)&stub, sizeof(P) * MAX_N * MAX_N * MAX_D);
	
	hipDeviceSynchronize();
			
	CUDA_SAFE_CALL( hipGetLastError() );
	
	for(int z = 0; z < BigCnt; ++z){
		for(int i = 0; i < n; ++i)graphptr[i] = 0;
		
		rnd_init(time(NULL) + z * 117);
		
		if (!generate_random_graph(n, d)) {
			printf("Invalid parameter.\n");
			continue;
		}
		
		for (int i = 0; i < n; ++i) {
			for (int j = i + 1; j < n; ++j) {
				if (G[i][j]){
					graph[i][graphptr[i]].to = j;
					graph[i][graphptr[i]++].dist = W[getI_C(i)][getI_C(j)];
					
					graph[j][graphptr[j]].to = i;
					graph[j][graphptr[j]++].dist = W[getI_C(i)][getI_C(j)];
				}
			}
		}
		
		for(int zz = 0; zz < smallCnt; ++zz){
			P v;
			int dim = treeDim_C(&v);
			
			
			hipMemcpy(WD, W, sizeof(int) * MAX_C * MAX_C, hipMemcpyHostToDevice);
			hipMemcpy(ND, N, sizeof(int) * MAX_C, hipMemcpyHostToDevice);
			hipMemcpy(graphD, graph, sizeof(P) * MAX_N * MAX_D, hipMemcpyHostToDevice);
			
			hipDeviceSynchronize();
			
			CUDA_SAFE_CALL( hipGetLastError() );
			
			initMinVArr<<<(n + BS - 1) / BS, BS>>>(n, minVarr);
			
			hipDeviceSynchronize();
			
			CUDA_SAFE_CALL( hipGetLastError() );
			
			hipDeviceSynchronize();
			
			CUDA_SAFE_CALL( hipGetLastError() );
			
			int ss =1;
			
			mm_main<<<1, n>>>(n, d, c, WD, ND, v, graphD, minVarr, stub);
			
			//mm_main<<<1, 1>>>(n, d, c, WD, ND, v, graphD, minVarr, stub);
			
			hipDeviceSynchronize();
			
			CUDA_SAFE_CALL( hipGetLastError() );
			
			getV<<<1,1>>>(n, d, minVarr, stub);
			
			P* stubAns = (P*)malloc(sizeof(P) * MAX_N * MAX_D);
			
			hipMemcpy(stubAns, stub, sizeof(P) * MAX_N * MAX_D, hipMemcpyDeviceToHost);
			
			int minVAns;
			
			hipMemcpy(&minVAns, minVarr, sizeof(int), hipMemcpyDeviceToHost);
			
			if(minVAns < dim){
				for(int i = 0; i < n; ++i){
					for(int j = 0; j < d; ++j){
						graph[i][j] = stubAns[i * MAX_D + j];
					}
				}
			}
			else{
				if(dim < allMin){
					allMin = dim;
					move();
				}
				free(stubAns);
				break;
			}
			free(stubAns);
		}
		if(allMin == INF)move();
	}
	
	for(int i = 0; i < n * d / 2; ++i){
		printf("%d %d\n", ans[i * 2], ans[i * 2 + 1]);
	}
	
	//printf("%d\n", allMin);
	
	return 0;
}
